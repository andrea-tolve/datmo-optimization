#include "hip/hip_runtime.h"
#include "cluster_cuda.hpp"

const int d = 25;
const int x_row = 1000;
const float step = (3.14 / 2) / d;
const int MAX_VAL = 10000;
const int MIN_VAL = -10000;

void memAllocate() {
    hipMalloc((void**)&d_idmaxth, sizeof(int));
    hipMalloc((void**)&d_x, sizeof(double) * x_row * 2);
    hipMalloc((void**)&d_row, sizeof(int));
    hipMalloc((void**)&d_max, sizeof(double) * 2 * d);
    hipMalloc((void**)&d_min, sizeof(double) * 2 * d);
    hipMalloc((void**)&d_coef, sizeof(double) * 12);
    hipMalloc((void**)&d_q, sizeof(double) * 2 * d);
    hipHostMalloc(&h_coef, sizeof(double) * 12);
    hipHostMalloc(&h_x, sizeof(double) * x_row * 2);
    hipStreamCreate(&s1); 
    hipStreamCreate(&s2);
    hipblasCreate(&handle);

    hipblasStatus_t stat;
    stat = hipblasSetStream(handle, s1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "errore nell'associazione tra handle e s1\n" << stat << std::endl;;
    }

    // specify that the pointer must be passed by reference to the device
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
}

void memFree() {
    hipFree(d_x);
    hipFree(d_row);
    hipFree(d_max);
    hipFree(d_min);
    hipFree(d_coef);
    hipFree(d_q);
    hipFree(&d_idmaxth);
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipblasDestroy(handle);
}

/*
    atomic operations for double; by default CUDA does not support 
    these operations on double types, but they can be implemented
    using cast to long long int
*/
__device__ void atomicMaxDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(fmax(val, __longlong_as_double(assumed)))); 
    } while (assumed != old); 
}

__device__ void atomicMinDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(fmin(val, __longlong_as_double(assumed)))); 
    } while (assumed != old); 
}

__device__ void atomicAddDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
}

/*
    function to compute the matrix-vector product (X * e1/e2)
    and to store the maximum and minimum values of c1 and c2
*/
__device__ void mulMatrix(double* c1, double* c2, double* max, double* min, double th, double* x, int col, int idx, int idy) {
    double e0, e1, e2, e3;
    double cos_th = cos(th);
    double sin_th = sin(th);

    e0 = cos_th;
    e1 = sin_th;
    e2 = -sin_th;
    e3 = cos_th;

    double x_idy = x[idy];
    double x_idy_row = x[idy + col];
    double sum;

    sum = x_idy * e0 + x_idy_row * e1; 
    c1[idy] = sum;

    atomicMaxDouble(&max[idx * 2], sum);
    atomicMinDouble(&min[idx * 2], sum);

    sum = x_idy * e2 + x_idy_row * e3;
    c2[idy] = sum;

    atomicMaxDouble(&max[idx * 2 + 1], sum);
    atomicMinDouble(&min[idx * 2 + 1], sum);


}

__global__ void calcQKernel(double* x, int* row, double* max, double* min, double* Q) {
    int idx = blockIdx.x;
    int idy = threadIdx.x;
    double th = step * idx;
    int r = row[0];
    if (idy == 0) {
        max[idx * 2] = MIN_VAL;
        max[idx * 2 + 1] = MIN_VAL;
        min[idx * 2] = MAX_VAL;
        min[idx * 2 + 1] = MAX_VAL;
    }

    extern __shared__ double sharedMem[];

    double* c1 = &sharedMem[0];
    double* c2 = &sharedMem[r];
    double* c1max = &sharedMem[2 * r];
    double* c2max = &sharedMem[3 * r];
    double* c1min = &sharedMem[4 * r];
    double* c2min = &sharedMem[5 * r];
   
    mulMatrix(c1, c2, max, min, th, x, r, idx, idy);
    __syncthreads();

    c1max[idy] = max[idx * 2] - c1[idy];
    c1min[idy] = c1[idy] - min[idx * 2];
    c2max[idy] = max[idx * 2 + 1] - c2[idy];
    c2min[idy] = c2[idy] - min[idx * 2 + 1];

    __syncthreads();

    //compute squared norm
    __shared__ double c1maxdata, c1mindata, c2maxdata, c2mindata, b; 
    if (idy == 0) {
        c1maxdata = 0;
        c1mindata = 0;
        c2maxdata = 0;
        c2mindata = 0;
        b = 0;
    }

    atomicAddDouble(&c1maxdata, c1max[idy] * c1max[idy]);
    atomicAddDouble(&c1mindata, c1min[idy] * c1min[idy]);
    atomicAddDouble(&c2maxdata, c2max[idy] * c2max[idy]);
    atomicAddDouble(&c2mindata, c2min[idy] * c2min[idy]);
    __syncthreads();


    double ma, mi, div, d0 = 0.001, val1, val2;

    bool cond1 = c1maxdata >= c1mindata;
    val1 = cond1 * c1min[idy] + !cond1 * c1max[idy]; 

    bool cond2 = c2maxdata >= c2mindata;
    val2 = cond2 * c2min[idy] + !cond2 * c2max[idy]; 


    mi = fmin(val1, val2);
    ma = fmax(mi, d0);
    div = 1 / ma;
    atomicAddDouble(&b, div);

    __syncthreads();

    //reuse the max array to store matrix Q
    if (idy == 0) {
        Q[idx * 2] = th;
        Q[idx * 2 + 1] = b;
    }

    
}


void launchKernelCuda(const double* X,const int num_point, double coef[]) {

    hipMemcpyAsync(d_row, &num_point, sizeof(int), hipMemcpyHostToDevice, s1);
    hipMemcpyAsync(d_x, X, sizeof(double) * num_point * 2, hipMemcpyHostToDevice, s1);
    int sharedMemorySize = 6 * num_point * sizeof(double);

    calcQKernel << <d, num_point, sharedMemorySize, s1 >> > (d_x, d_row, d_max, d_min, d_q);
   
    int idmax;
	
    //check only column b, starting from the second element and using stride = 2 (skip the element in between)
    hipblasIdamax(handle, d, d_q+1, 2, &idmax); //indice che parte da 1
 
    idmax -= 1;
    double th = step * idmax, h_min[2], h_max[2];

    hipMemcpyAsync(h_min, d_min + idmax * 2, sizeof(double) * 2, hipMemcpyDeviceToHost, s1);
    hipMemcpyAsync(h_max, d_max + idmax * 2, sizeof(double) * 2, hipMemcpyDeviceToHost, s2);

    hipStreamSynchronize(s1);
    hipStreamSynchronize(s2);

    coef[0] = cos(th);
    coef[1] = sin(th);
    coef[2] = h_min[0];
    coef[3] = -sin(th);
    coef[4] = coef[0];
    coef[5] = h_min[1];
    coef[6] = coef[0];
    coef[7] = coef[1];
    coef[8] = h_max[0];
    coef[9] = coef[3];
    coef[10] = coef[0];
    coef[11] = h_max[1];


}
